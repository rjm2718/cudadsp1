#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <arpa/inet.h>
#include <chrono>

#include "rtpdsp.h"
#include "ulaw.h"


int16_t search(int16_t val, int16_t *table, int16_t size)
{
    int16_t i;

    for (i = 0; i < size; i++) {
        if (val <= *table++)
            return (i);
    }
    return (size);
}

uint8_t pcm2ulaw(int16_t pcm_val)	/* 2's complement (16-bit range) */
{
    int16_t mask;
    int16_t seg;
    uint8_t uval;

    /* Get the sign and the magnitude of the value. */
    pcm_val = pcm_val >> 2;
    if (pcm_val < 0) {
        pcm_val = -pcm_val;
        mask = 0x7F;
    } else {
        mask = 0xFF;
    }
    if ( pcm_val > CLIP ) pcm_val = CLIP;		/* clip the magnitude */
    pcm_val += (BIAS >> 2);

    /* Convert the scaled magnitude to segment number. */
    seg = search(pcm_val, seg_uend, 8);

    /*
    * Combine the sign, segment, quantization bits;
    * and complement the code word.
    */
    if (seg >= 8)		/* out of range, return maximum value. */
        return (unsigned char) (0x7F ^ mask);
    else {
        uval = (unsigned char) (seg << 4) | ((pcm_val >> (seg + 1)) & 0xF);
        return (uval ^ mask);
    }

}


void add_sine_wave(int16_t *buffer, int length, double amplitude, double frequency, double sample_rate) {
    const double two_pi = 2.0 * M_PI;
    for (int i = 0; i < length; i++) {
        double t = static_cast<double>(i) / sample_rate; // Time for current sample
        buffer[i] += static_cast<int16_t>(amplitude * sin(two_pi * frequency * t));
    }
}

void mk_rtp_packet(rtp_packet *pkt, uint32_t ssrc) {

    // Set the header fields
    pkt->header[0] = 0x80; // Version 2, no padding, no extensions, CC=0
    pkt->header[1] = 0; // payload type ulaw

    // increment sequence number
    uint16_t *seqnum = (uint16_t*)&pkt->header[2];
    *seqnum++;

    uint32_t ssrc_network = htonl(ssrc);
    memcpy(&pkt->header[8], &ssrc_network, sizeof(uint32_t)); // Copy SSRC
    pkt->ssrc = ssrc;

    // simple sine wave composite
    int16_t buffer[RTP_PAYLOAD_LEN];
    memset(buffer, 0, sizeof(buffer));
    add_sine_wave(buffer, RTP_PAYLOAD_LEN, 4000, 32, 4096);

    double freq = 100 + ssrc % 1000;
    add_sine_wave(buffer, RTP_PAYLOAD_LEN, 8000, freq, 4096);

    // convert to ulaw
    for (int i = 0; i < RTP_PAYLOAD_LEN; i++) {
        // printf("%d ", buffer[i]);
        pkt->payload[i] = pcm2ulaw(buffer[i]);
    }
    // printf("\n");
}

void printPktSpctrm(void* ps, int n) {
    char fn[100];
    sprintf(fn, "spectrum-%d.csv", n);
    FILE *file = fopen(fn, "w");
    if (!file) {
        perror("Failed to open file");
        return;
    }
    pktspectrum s = ((pktspectrum *) ps)[n];
    for (int i = 0; i < FFT::output_length; i++) {
        fprintf(file, "%.1f %.1f\n", s.spectrum[i].x, s.spectrum[i].y);
    }
    fclose(file);
    printf("wrote data to %s for ssrc %d: 8, %.1f\n", fn, s.ssrc, (100+n%1000)/32.0*8.0);
}

void processPackets(void* pktbuf_h, void* pktbuf_d, void* pktspcbuf_h, void* pktspcbuf_d, int n_packets, int n_iterations) {

    for (int i = 0; i < n_iterations; i++) {

        CUDA_ERR_CHK( hipMemcpy(pktbuf_d, pktbuf_h, sizeof(rtp_packet) * n_packets, hipMemcpyHostToDevice) );

        // Timing setup
        // hipEvent_t start, stop;
        // hipEventCreate(&start);
        // hipEventCreate(&stop);
        // hipEventRecord(start, 0);

        kernel_dsp<<<n_packets, THREADS_PER_BLOCK, FFT::shared_memory_size>>>((rtp_packet*)pktbuf_d, (pktspectrum*)pktspcbuf_d, n_packets);

        // hipEventRecord(stop, 0);

        CUDA_ERR_CHK(hipGetLastError()); // Check kernel launch errors

        CUDA_ERR_CHK(hipMemcpy(pktspcbuf_h, pktspcbuf_d, sizeof(pktspectrum) * n_packets, hipMemcpyDeviceToHost));

        // Wait for the event to complete
        // hipEventSynchronize(stop);

        // Calculate elapsed time in milliseconds
        // float elapsedTime;
        // hipEventElapsedTime(&elapsedTime, start, stop);
        // std::cout << "Kernel execution time: " << elapsedTime << " ms\n";
    }
    
    CUDA_ERR_CHK(hipDeviceSynchronize());
}

int main(int argc, char** argv) {

    int n_iterations = 4;
    if (argc > 1) {
        n_iterations = std::stoi(argv[1]);
    }

    int NUM_PACKETS = 20000;

    // allocate host buffer and generate test packets
    void* pktbuf_h = malloc(sizeof(rtp_packet) * NUM_PACKETS);
    void* pbuf = pktbuf_h;
    for (int i = 0; i < NUM_PACKETS; i++, pbuf += sizeof(rtp_packet)) {
        uint32_t ssrc = i;
        mk_rtp_packet((rtp_packet*)pbuf, ssrc);
    }

    // rtp_packet p10 = ((rtp_packet*)pktbuf_h)[10];
    // for (int i = 0; i < RTP_PAYLOAD_LEN; i++) {
    //     printf("%d\n", p10.payload[i]);
    // }
    // return 0;

    void* pktbuf_d;
    CUDA_ERR_CHK( hipMalloc(&pktbuf_d, sizeof(rtp_packet) * NUM_PACKETS) );

    // results buffer
    void* pktspcbuf_h = malloc(sizeof(pktspectrum) * NUM_PACKETS);
    void* pktspcbuf_d;
    CUDA_ERR_CHK( hipMalloc(&pktspcbuf_d, sizeof(pktspectrum) * NUM_PACKETS) );

    assert(!FFT::requires_workspace);

    printf("num packets %d\n", NUM_PACKETS);
    printf("fft input_length %d\n", FFT::input_length);
    printf("fft output_length %d\n", FFT::output_length);
    printf("block size %d, elements per thread %d\n", ELEMENTS_PER_THREAD, THREADS_PER_BLOCK);
    printf("per-SM shared memory size %d KB\n", FFT::shared_memory_size/1024);
    printf("hostToDevice copy %d MB\n", sizeof(rtp_packet) * NUM_PACKETS/1024/1024);
    printf("deviceToHost copy %d MB\n", sizeof(pktspectrum) * NUM_PACKETS/1024/1024);

    auto start_time = std::chrono::high_resolution_clock::now();

    processPackets(pktbuf_h, pktbuf_d, pktspcbuf_h, pktspcbuf_d, NUM_PACKETS, n_iterations);

    auto end_time = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time).count();
    auto us_per_packet = (double)duration / NUM_PACKETS / n_iterations * 1000.0;
    printf("\niterations: %d\n", n_iterations);
    printf("total time: %d ms (%.1f ms per iteration, %.1f μs per packet)\n", duration, (float)duration/n_iterations, us_per_packet);



    // printPktSpctrm(pktspcbuf_h, 10);
    // printPktSpctrm(pktspcbuf_h, 150);
    // printPktSpctrm(pktspcbuf_h, 450);

    return 0;
}